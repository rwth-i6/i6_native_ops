#ifdef RNNT_ENABLE_GPU

#include <ATen/cuda/HIPContext.h>

#include "gpu_rnnt.h"
#include "gpu_workspace_manager.h"

#endif

#include <torch/extension.h>

#include "cpu_rnnt.h"
#include "cpu_workspace_manager.h"
#include "options.h"

int cpu_monotonic_rnnt(torch::Tensor& acts, torch::Tensor& labels, torch::Tensor& input_lengths,
                       torch::Tensor& label_lengths, torch::Tensor& costs, torch::Tensor& grads, int blank_label,
                       int num_threads) {
    TORCH_CHECK(acts.type().scalarType() == torch::ScalarType::Float);

    int B = labels.size(0);
    int V = acts.size(1);

    RNNTOptions options;
    options.loc = RNNT_CPU;
    options.blank_label = blank_label;
    options.num_threads = num_threads;

    CpuRNNTWorkspaceManager<float> workspace_manager(acts.data_ptr<float>(), labels.data_ptr<int>(),
                                                     static_cast<int>(B), input_lengths.data_ptr<int>(),
                                                     label_lengths.data_ptr<int>(), static_cast<int>(V));
    auto rnnt_status = workspace_manager.create_workspace();

    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "cpu_rnnt error in create_workspace");

    CpuRNNTComputer<float> rnnt_computer(workspace_manager, options.blank_label, options.num_threads);

    rnnt_status = rnnt_computer.cost_and_grad(costs.data_ptr<float>(), grads.data_ptr<float>());
    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "cpu_rnnt error in rnnt_computer");

    workspace_manager.free_workspace();

    return rnnt_status;
}

int cpu_monotonic_rnnt_align_restrict(torch::Tensor& acts, torch::Tensor& labels, torch::Tensor& input_lengths,
                                      torch::Tensor& label_lengths, torch::Tensor& alignment,
                                      int max_shift_from_alignment, torch::Tensor& costs, torch::Tensor& grads,
                                      int blank_label, int num_threads) {
    TORCH_CHECK(acts.type().scalarType() == torch::ScalarType::Float);

    int B = labels.size(0);
    int V = acts.size(1);

    RNNTOptions options;
    options.loc = RNNT_CPU;
    options.blank_label = blank_label;
    options.num_threads = num_threads;

    CpuRNNTWorkspaceManager<float> workspace_manager(acts.data_ptr<float>(), labels.data_ptr<int>(),
                                                     static_cast<int>(B), input_lengths.data_ptr<int>(),
                                                     label_lengths.data_ptr<int>(), static_cast<int>(V));
    auto rnnt_status = workspace_manager.create_workspace();

    workspace_manager.restrict_to_alignment(alignment.data_ptr<int>(), max_shift_from_alignment, blank_label);

    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "cpu_rnnt error in create_workspace");

    CpuRNNTComputer<float> rnnt_computer(workspace_manager, options.blank_label, options.num_threads);

    rnnt_status = rnnt_computer.cost_and_grad(costs.data_ptr<float>(), grads.data_ptr<float>());
    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "cpu_rnnt error in rnnt_computer");

    workspace_manager.free_workspace();

    return rnnt_status;
}

#ifdef RNNT_ENABLE_GPU

int gpu_monotonic_rnnt(torch::Tensor& acts, torch::Tensor& labels, torch::Tensor& input_lengths,
                       torch::Tensor& label_lengths, torch::Tensor& costs, torch::Tensor& grads, int blank_label,
                       int num_threads) {
    TORCH_CHECK(acts.type().scalarType() == torch::ScalarType::Float);
    TORCH_CHECK(acts.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(labels.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(input_lengths.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(label_lengths.type().is_cuda(), "acts must be a CUDA tensor");

    int B = labels.size(0);
    int V = acts.size(1);

    RNNTOptions options;
    options.loc = RNNT_GPU;
    options.blank_label = blank_label;
    options.stream = at::cuda::getCurrentCUDAStream();
    options.num_threads = num_threads;

    GpuRNNTWorkspaceManager<float> workspace_manager(acts.data_ptr<float>(), labels.data_ptr<int>(),
                                                     static_cast<int>(B), input_lengths.data_ptr<int>(),
                                                     label_lengths.data_ptr<int>(), static_cast<int>(V));

    auto rnnt_status = workspace_manager.create_workspace();

    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "gpu_rnnt error in create_workspace");

    GpuRNNTComputer<float> rnnt_computer(workspace_manager, options.blank_label, options.stream);
    rnnt_status = rnnt_computer.cost_and_grad(costs.data_ptr<float>(), grads.data_ptr<float>());
    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "gpu_rnnt error in rnnt_computer");

    workspace_manager.free_workspace();

    return rnnt_status;
}

int gpu_monotonic_rnnt_align_restrict(torch::Tensor& acts, torch::Tensor& labels, torch::Tensor& input_lengths,
                                      torch::Tensor& label_lengths, torch::Tensor& alignment,
                                      int max_shift_from_alignment, torch::Tensor& costs, torch::Tensor& grads,
                                      int blank_label, int num_threads) {
    TORCH_CHECK(acts.type().scalarType() == torch::ScalarType::Float);
    TORCH_CHECK(acts.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(labels.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(input_lengths.type().is_cuda(), "acts must be a CUDA tensor");
    TORCH_CHECK(label_lengths.type().is_cuda(), "acts must be a CUDA tensor");

    int B = labels.size(0);
    int V = acts.size(1);

    RNNTOptions options;
    options.loc = RNNT_GPU;
    options.blank_label = blank_label;
    options.stream = at::cuda::getCurrentCUDAStream();
    options.num_threads = num_threads;

    GpuRNNTWorkspaceManager<float> workspace_manager(acts.data_ptr<float>(), labels.data_ptr<int>(),
                                                     static_cast<int>(B), input_lengths.data_ptr<int>(),
                                                     label_lengths.data_ptr<int>(), static_cast<int>(V));

    auto rnnt_status = workspace_manager.create_workspace();

    workspace_manager.restrict_to_alignment(alignment.data_ptr<int>(), max_shift_from_alignment, blank_label);

    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "gpu_rnnt error in create_workspace");

    GpuRNNTComputer<float> rnnt_computer(workspace_manager, options.blank_label, options.stream);
    rnnt_status = rnnt_computer.cost_and_grad(costs.data_ptr<float>(), grads.data_ptr<float>());
    TORCH_CHECK(rnnt_status == RNNT_STATUS_SUCCESS, "gpu_rnnt error in rnnt_computer");

    workspace_manager.free_workspace();

    return rnnt_status;
}
#endif

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("cpu_monotonic_rnnt", &cpu_monotonic_rnnt, "Monotonic RNNT CPU version");
    m.def("cpu_monotonic_rnnt_align_restrict", &cpu_monotonic_rnnt_align_restrict,
          "Alignment-restricted monotonic RNNT CPU version");
#ifdef RNNT_ENABLE_GPU
    m.def("gpu_monotonic_rnnt", &gpu_monotonic_rnnt, "Monotonic RNNT GPU version");
    m.def("gpu_monotonic_rnnt_align_restrict", &gpu_monotonic_rnnt_align_restrict,
          "Alignment-restricted monotonic RNNT GPU version");
#endif
}
